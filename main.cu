#include"binary.h"
#include"get_input.cu"
#include"init_conf.cu"
#include"evolve.cu"
#include"out_conf.cu"

int main(void)
{
    void Get_Input_Parameters(char *fnin, char *fnout);
    void Init_Conf(void);
    void Read_Restart(void);
    void Evolve(void);
    void Output_Conf(int );

    char finput[15] = "bin1ary";
    char fnin[30], fnout[30];

    FILE *fp;

    if (!(fp = fopen(finput, "r"))) {
	printf("File:%s could not be opened\n", finput);
	exit(EXIT_FAILURE);
    }
    if (fscanf(fp, "%s", fnin) == 1) {
	printf("Input Parameters Filename:%s\n", fnin);
    }
    if (fscanf(fp, "%s", fnout) == 1) {
	printf("Output Parameters Filename:%s\n", fnout);
    }
    if (!(fpout = fopen(fnout, "w"))) {
	printf("File:%s could not be opened\n", fnout);
	exit(EXIT_FAILURE);
    }
    fclose(fp);

    Get_Input_Parameters(fnin, fnout);

    //comp   = (hipDoubleComplex*)malloc( (sizeof (hipDoubleComplex)*nx*ny*nz) );
    //dfdc   = (hipDoubleComplex*)malloc( (sizeof (hipDoubleComplex)*nx*ny*nz) );
   
    

    hipMalloc ((void **)&comp_d, nx*ny*nz*sizeof(hipDoubleComplex));
    hipMalloc ((void **)&dfdc_d, nx*ny*nz*sizeof(hipDoubleComplex));
    
    
    
    one_by_nxnynz = 1.0 / (double) (nx * ny * nz);
    blocks=(nx*ny*nz)/1024;
    
    hipfftPlan3d(&plan, nx, ny, nz, HIPFFT_Z2Z);


    
	Init_Conf();
   

    //hipMemcpy(dfdc_d,dfdc,sizeof(hipDoubleComplex)*nx*ny*nz,hipMemcpyHostToDevice);
    hipMemcpy(dfdc_d,dfdc,sizeof(hipDoubleComplex)*nx*ny*nz,hipMemcpyHostToDevice);
    
    Evolve();


    fclose(fpout);

    hipfftDestroy(plan);
    hipFree(comp_d);
    hipFree(dfdc_d);
    
    return 0;
}
